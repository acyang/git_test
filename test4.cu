#include "hip/hip_runtime.h"
//�h�϶�, �h����� (�ϥΰj��)
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <ctime>

//----------------------------------------------
//�V�q�[�k���B��֤� (GPU) **�禡�e�[ __global__ �Y���֤�, �֤ߥu�Ǧ^ void**
__global__ void gpu_add(float* c, float* a, float* b, int n){
        int j=blockIdx.x*blockDim.x+threadIdx.x;
        int m=gridDim.x*blockDim.x;
        for(int k=j; k<n; k+=m){
                c[k]=a[k]+b[k];
        }
}


//----------------------------------------------
//�V�q�[�k���@��禡 (Host)
void host_add(float* c, float* a, float* b, int n){
        for(int k=0; k<n; k++){
                c[k]=a[k]+b[k];
        }
}


//----------------------------------------------
//�p��~�t�Ϊ��禡
double diff(float* a, float* b, int n){
        double s=0, r=0;
        for(int k=0; k<n; k++){
                double w=a[k]-b[k];
                s+=w*w;
                r+=a[k]*a[k];
        }
        return sqrt(s/r); //�۹�~�t
}

//----------------------------------------------
//�ɶ���� (�Ǧ^���:�d�����@��)
double ms_time(){
        return (double)clock()/CLOCKS_PER_SEC*1000.0;
}

//----------------------------------------------
//�D�{��
int main(){
        //�]�w�V�q�j�p
        int n=1024*1024;
        int size=n*sizeof(float);

        //����P�϶��]�w
        int block=256;    //blockDim (�C�Ӱ϶��㦳���������)
        int grid=30;     //gridDim  (�C�Ӻ���㦳���϶���)

        //�]�w�I�s���� (���q�����į�)
        int loop=100;

        //�t�m�D���O����
        float *a,*b,*c,*d;
        a=(float*)malloc(size);
        b=(float*)malloc(size);
        c=(float*)malloc(size);
        d=(float*)malloc(size);

        //�]�w�üƪ���J�V�q
        srand(time(0));
        for(int k=0; k<n; k++){
                a[k]=(float)rand()/RAND_MAX*2-1;
                b[k]=(float)rand()/RAND_MAX*2-1;
        }

        //�t�m��ܥd�O����
        float  *ga,*gb,*gc;
        hipMalloc((void**)&ga, size);
        hipMalloc((void**)&gb, size);
        hipMalloc((void**)&gc, size);

        //���J�V�q a,b ����ܥd�O���餤
        hipMemcpy(ga, a, size, hipMemcpyHostToDevice);
        hipMemcpy(gb, b, size, hipMemcpyHostToDevice);

        //---- part 1 : ���q��T�� --------

        //�I�s�֤ߨӹB�� (GPU)
        gpu_add<<<grid, block>>>(gc, ga, gb, n);

        //�I�s�@���ƨӹB�� (Host)
        host_add(d, a, b, n);

        //��p�⵲�G�s�^�D��
        hipMemcpy(c, gc, size, hipMemcpyDeviceToHost);

        //�����̮t��
        printf("vector add N(%d) elements, diff = %g\n", n, diff(c,d,n));



        //---- part 2 : ���q�į� --------

        //���q GPU �֤߮į�
        double gpu_dt = ms_time();
        for(int w=0; w<loop; w++){
                gpu_add<<<grid, block>>>(gc, ga, gb, n);
                hipDeviceSynchronize();  //�קK�֤߰��椣����
        }
        gpu_dt = (ms_time()-gpu_dt)/loop; //�����ɶ�


        //���q Host ��Ʈį�
        double host_dt = ms_time();
        for(int w=0; w<loop; w++){
                host_add(d, a, b, n);
        }
        host_dt = (ms_time()-host_dt)/loop; //�����ɶ�


        //��X��������ɶ�
        printf("host time: %g ms\n", host_dt);
        printf("gpu  time: %g ms\n", gpu_dt);


        //����D���O����
        free(a);
        free(b);
        free(c);
        free(d);

        //������ܥd�O����
        hipFree(ga);
        hipFree(gb);
        hipFree(gc);

        return 0;
}